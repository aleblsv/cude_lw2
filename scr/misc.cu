//
// Created by Alex on 3/7/2017.
//

#include "config.h"
#include "misc.h"

/* Private define ------------------------------------------------------------*/
/* Private typedef -----------------------------------------------------------*/
/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
/* Private variables ---------------------------------------------------------*/
static MISC_Bl_Size_TypeDef sMISC_Block_Size = {256, 16};
/* ---------------------------------------------------------------------------*/

/**
 *@brief Check device compute capability
 *@param
 *@retval None
 */
void MISC_Check_Device(void)
{
    int devID;
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr,
                "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error,
               __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major,
               deviceProp.minor);
    }

    // Use a larger block size for Fermi and above
    sMISC_Block_Size.Bl_2d = (deviceProp.major < 2) ? 16 : 32;
    sMISC_Block_Size.Bl_1d = sMISC_Block_Size.Bl_2d * sMISC_Block_Size.Bl_2d;
}

/**
 *@brief Check device compute capability
 *@param
 *@retval None
 */
MISC_Bl_Size_TypeDef MISC_Get_Block_Size(void)
{
    return sMISC_Block_Size;
}

/**
 *@brief Print z vector
 *@param
 *@retval None
 */
void MISC_Print_Z_Vec(Tp_Z_Vec_TypeDef Z_Vec)
{
    for (int i = 0; i < Z_Vec.Size; i++)
    {
        printf("label:%d, is_proto:%d, num_of_features:%d ->[",
               Z_Vec.pElements[i].Label,
               Z_Vec.pElements[i].IsProto,
               Z_Vec.pElements[i].Size);
        for (int j = 0; j < Z_Vec.pElements[i].Size; j++)
        {
            printf("%.2f ", Z_Vec.pElements[i].Feature_Arr[j]);
        }
        printf("]\n");
    }
}


