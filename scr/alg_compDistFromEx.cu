#include "hip/hip_runtime.h"
//
// Created by Alex on 3/7/2017.
//

#include <stdlib.h>
#include <stdio.h>

#include "alg_compDistFromEx.h"
#include "types.h"
#include "misc.h"
#include "mat.h"
#include "dist.h"

/* Private define ------------------------------------------------------------*/
/* Private typedef -----------------------------------------------------------*/
/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
/* Private variables ---------------------------------------------------------*/
/* ---------------------------------------------------------------------------*/

/**
 *@brief  GPU - kernel, calculate distances of Z and U vectors building D matrix
 *@param
 *@retval None
 */
__global__ void ALG_compDistFromEx_Kernel(Tp_intVec_TypeDef Z_Vec, Tp_intVec_TypeDef U_Vec, Tp_fMat_TypeDef D_Mat)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < D_Mat.Height && col < D_Mat.Width)
    {
        MAT_SetElement(D_Mat, row, col, DIST_Calc(Z_Vec.pElements[row], U_Vec.pElements[col]));
    }
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_compDistFromEx_Launch(const Tp_intVec_TypeDef Z_Vec, const Tp_intVec_TypeDef U_Vec, Tp_fMat_TypeDef D_Mat)
{
    StopWatchInterface *timer = NULL;

    Tp_intVec_TypeDef d_Z_Vec;
    Tp_intVec_TypeDef d_U_Vec;
    Tp_fMat_TypeDef d_D_Mat;
    size_t Size;
    MISC_Bl_Size_TypeDef DimBlck = MISC_Get_Block_Size();

    printf("\nGPU kernel compDistFromEx - Start\n");
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    d_Z_Vec = Z_Vec;
    Size = d_Z_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_Z_Vec.pElements, Size));
    checkCudaErrors(hipMemcpy(d_Z_Vec.pElements, Z_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_U_Vec = U_Vec;
    Size = d_U_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_U_Vec.pElements, Size));
    checkCudaErrors(hipMemcpy(d_U_Vec.pElements, U_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_D_Mat = D_Mat;
    Size = d_D_Mat.Width * d_D_Mat.Height * sizeof(float);
    checkCudaErrors(hipMalloc(&d_D_Mat.pElements, Size));
    checkCudaErrors(hipMemcpy(d_D_Mat.pElements, D_Mat.pElements, Size, hipMemcpyHostToDevice));

    // Invoke kernel
    dim3 dimBlock(DimBlck.Bl_2d, DimBlck.Bl_2d);
    dim3 dimGrid((d_D_Mat.Width + dimBlock.x - 1) / dimBlock.x, (d_D_Mat.Height + dimBlock.y - 1) / dimBlock.y);
    ALG_compDistFromEx_Kernel << < dimGrid, dimBlock >> > (d_Z_Vec, d_U_Vec, d_D_Mat);
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(D_Mat.pElements, d_D_Mat.pElements, Size, hipMemcpyDeviceToHost));

//    Free device memory
    checkCudaErrors(hipFree(d_Z_Vec.pElements));
    checkCudaErrors(hipFree(d_U_Vec.pElements));
    checkCudaErrors(hipFree(d_D_Mat.pElements));

    sdkStopTimer(&timer);
    printf("GPU kernel - Complete, time:%fms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_compDistFromEx_Test(void)
{
    int z_arr[] = {1, 3, 4, 5, 7};
    int u_arr[] = {8, 9, 10};
    Tp_intVec_TypeDef h_Z_Vec;
    Tp_intVec_TypeDef h_U_Vec;
    Tp_fMat_TypeDef h_D_Mat;
    size_t Size;

    h_Z_Vec.pElements = z_arr;
    h_Z_Vec.Size = sizeof(z_arr) / sizeof(z_arr[0]);
    h_U_Vec.pElements = u_arr;
    h_U_Vec.Size = sizeof(u_arr) / sizeof(u_arr[0]);

    h_D_Mat.Height = h_Z_Vec.Size;
    h_D_Mat.Width = h_U_Vec.Size;
    Size = h_D_Mat.Height * h_D_Mat.Width * sizeof(float);
    h_D_Mat.pElements = (float *) malloc(Size);
    if (h_D_Mat.pElements == NULL)
    {
        printf("Can't allocate memory\n");
        return;
    }
    MAT_SetElementAll(h_D_Mat, 0.0);
    MAT_PrintVecInt(h_Z_Vec);
    MAT_PrintVecInt(h_U_Vec);
    MAT_PrintMat(h_D_Mat);
    ALG_compDistFromEx_Launch(h_Z_Vec, h_U_Vec, h_D_Mat);
    MAT_PrintMat(h_D_Mat);
    free(h_D_Mat.pElements);
}