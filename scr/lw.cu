#include "hip/hip_runtime.h"
//
// Created by Alex on 15/6/2018.
//

#include <lw.h>

#include <assert.h>
#include <string.h>
#include <stdio.h>

#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "max_min.h"
#include "config.h"

/* Private define ------------------------------------------------------------*/

/* Private typedef -----------------------------------------------------------*/

/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
static void _LW_Launch_Min2(float *pV, int *pPsy, int m_len, int *pU, int M_len, int *pIndex_Out, float *pMin_Out);
/* Private variables ---------------------------------------------------------*/

/* ---------------------------------------------------------------------------*/

/**
 *@brief  GPU - kernel
 *@param
 *@retval None
 */
__global__ void
LW_Kernel_Min2(float *pMin_d, float *pV_d, int *pPsy_d, int m_len, int *pU_d, int M_len, int *pIndex_Out_d, int *mutex)
{
    int i_m = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    float minVal;

    if (i_m < m_len)
    {
        pMin_d[i_m] = MAX_MIN_INF;
        for (j = 0; j < M_len; j++)
        {
            if (pU_d[j] == 1)
            {
                if (pPsy_d[i_m] == j)
                {
                    pMin_d[i_m] = pV_d[i_m];
                    break;
                }
            }
        }
    }
    __syncthreads();

    find_minimum_index_kernel(pMin_d, &minVal, pIndex_Out_d, mutex, m_len);
}

/**
 *@brief  Pre-launch kernel function
 *@param
 *@retval None
 */
static void _LW_Launch_Min2(float *pV, int *pPsy, int m_len, int *pU, int M_len, int *pIndex_Out, float *pMin_Out)
{
    float *pMin_d;
    float *pV_d;
    int *pPsy_d;
    int *pU_d;
    int *pIndex_Out_d;
    int *d_mutex;

    // Allocate memory on  Device
    checkCudaErrors(hipMalloc((void **) &pMin_d, m_len * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &pV_d, m_len * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &pPsy_d, m_len * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &pU_d, M_len * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &pIndex_Out_d, sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_mutex, sizeof(int)));

    // Copy data from Host memory to Device memory
    checkCudaErrors(hipMemcpy(pV_d, pV, m_len * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pPsy_d, pPsy, m_len * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pU_d, pU, M_len * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_mutex, 0, sizeof(int)));

    int threadsPerBlock = CONFIG_THREADS_PER_BLOCK;
    int blocksPerGrid = (m_len + threadsPerBlock - 1) / threadsPerBlock;
    // launch kernel
    LW_Kernel_Min2 << < blocksPerGrid, threadsPerBlock >> >
                                       (pMin_d, pV_d, pPsy_d, m_len, pU_d, M_len, pIndex_Out_d, d_mutex);
    hipDeviceSynchronize();

    // Copy result from Device memory to Host memory
    checkCudaErrors(hipMemcpy(pIndex_Out, pIndex_Out_d, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pMin_Out, pMin_d, m_len * sizeof(float), hipMemcpyDeviceToHost));

    // Free Device memory
    checkCudaErrors(hipFree(pMin_d));
    checkCudaErrors(hipFree(pV_d));
    checkCudaErrors(hipFree(pPsy_d));
    checkCudaErrors(hipFree(pU_d));
    checkCudaErrors(hipFree(pIndex_Out_d));
    checkCudaErrors(hipFree(d_mutex));
}

/**
 *@brief  Enter Function
 *@param  pV: pointer to v array
 *@param  pPsy: pointer to psy array
 *@param  m_len: m length
 *@param  pU: pointer to u array
 *@param  M_len: M length
 *@param  pIndex_Out: Result, index of minimum value of v array
 *@retval None
 */
void LW_Calculate_Min2(float *pV, int *pPsy, int m_len, int *pU, int M_len, int *pIndex_Out)
{
    StopWatchInterface *timer = NULL;
    float ptMin[m_len];

    printf("\nGPU kernel - Start\n");
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    sdkStartTimer(&timer);
    _LW_Launch_Min2(pV, pPsy, m_len, pU, M_len, pIndex_Out, ptMin);
    sdkStopTimer(&timer);

    printf("GPU kernel - Complete, time:%fms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

/**
 *@brief  Test
 *@param  none
 *@retval None
 */
void LW_Test_Min2(void)
{
    int index = 0;
    float v_arr[9] = {4, 52, 11, 5, 6, 7, 8, 9, 10};
    int psy_arr[9] = {0, 2, 9, 9, 9, 9, 9, 9, 0};
    int u_arr[3] = {1, 0, 1};

    printf("Start Test\n");
    LW_Calculate_Min2(v_arr, psy_arr, 9, u_arr, 3, &index);
    printf("Stop Test, v[%d]=%f\n", index, v_arr[index]);
}

