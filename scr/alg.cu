#include "hip/hip_runtime.h"
//
// Created by Alex on 3/7/2017.
//

#include <stdlib.h>
#include <stdio.h>

#include "alg.h"
#include "config.h"
#include "types.h"
#include "misc.h"
#include "mat.h"

/* Private define ------------------------------------------------------------*/
/* Private typedef -----------------------------------------------------------*/
/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
/* Private variables ---------------------------------------------------------*/
/* ---------------------------------------------------------------------------*/

/**
 *@brief  GPU - kernel, calculate distance
 *@param
 *@retval None
 */
__host__ __device__ float ALG_Dist(int v1, int v2)
{
    //ToDo: Need to change to distance function
    return (float) (v1 + v2);
}

/**
 *@brief  GPU - kernel, calculate distances of Z and U vectors building D matrix
 *@param
 *@retval None
 */
__global__ void ALG_compDistFromEx_Kernel(Tp_intVec_TypeDef Z_Vec, Tp_intVec_TypeDef U_Vec, Tp_fMat_TypeDef D_Mat)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < D_Mat.Height && col < D_Mat.Width)
    {
        MAT_SetElement(D_Mat, row, col, ALG_Dist(Z_Vec.Elements[row], U_Vec.Elements[col]));
    }
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_compDistFromEx_Launch(const Tp_intVec_TypeDef Z_Vec, const Tp_intVec_TypeDef U_Vec, Tp_fMat_TypeDef D_Mat)
{
    StopWatchInterface *timer = NULL;

    Tp_intVec_TypeDef d_Z_Vec;
    Tp_intVec_TypeDef d_U_Vec;
    Tp_fMat_TypeDef d_D_Mat;
    size_t Size;
    MISC_Bl_Size_TypeDef DimBlck = MISC_Get_Block_Size();

    printf("\nGPU kernel compDistFromEx - Start\n");
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    d_Z_Vec = Z_Vec;
    Size = d_Z_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_Z_Vec.Elements, Size));
    checkCudaErrors(hipMemcpy(d_Z_Vec.Elements, Z_Vec.Elements, Size, hipMemcpyHostToDevice));

    d_U_Vec = U_Vec;
    Size = d_U_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_U_Vec.Elements, Size));
    checkCudaErrors(hipMemcpy(d_U_Vec.Elements, U_Vec.Elements, Size, hipMemcpyHostToDevice));

    d_D_Mat = D_Mat;
    Size = d_D_Mat.Width * d_D_Mat.Height * sizeof(float);
    checkCudaErrors(hipMalloc(&d_D_Mat.Elements, Size));
    checkCudaErrors(hipMemcpy(d_D_Mat.Elements, D_Mat.Elements, Size, hipMemcpyHostToDevice));

    // Invoke kernel
    dim3 dimBlock(DimBlck.Bl_2d, DimBlck.Bl_2d);
    dim3 dimGrid((d_D_Mat.Width + dimBlock.x - 1) / dimBlock.x, (d_D_Mat.Height + dimBlock.y - 1) / dimBlock.y);
    ALG_compDistFromEx_Kernel << < dimGrid, dimBlock >> > (d_Z_Vec, d_U_Vec, d_D_Mat);
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(D_Mat.Elements, d_D_Mat.Elements, Size, hipMemcpyDeviceToHost));

//    Free device memory
    checkCudaErrors(hipFree(d_Z_Vec.Elements));
    checkCudaErrors(hipFree(d_U_Vec.Elements));
    checkCudaErrors(hipFree(d_D_Mat.Elements));

    sdkStopTimer(&timer);
    printf("GPU kernel - Complete, time:%fms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_compDistFromEx_Test(void)
{
    int z_arr[] = {1, 3, 4, 5, 7};
    int u_arr[] = {8, 9, 10};
    Tp_intVec_TypeDef h_Z_Vec;
    Tp_intVec_TypeDef h_U_Vec;
    Tp_fMat_TypeDef h_D_Mat;
    size_t Size;

    h_Z_Vec.Elements = z_arr;
    h_Z_Vec.Size = sizeof(z_arr) / sizeof(z_arr[0]);
    h_U_Vec.Elements = u_arr;
    h_U_Vec.Size = sizeof(u_arr) / sizeof(u_arr[0]);

    h_D_Mat.Height = h_Z_Vec.Size;
    h_D_Mat.Width = h_U_Vec.Size;
    Size = h_D_Mat.Height * h_D_Mat.Width * sizeof(float);
    h_D_Mat.Elements = (float *) malloc(Size);
    if (h_D_Mat.Elements == NULL)
    {
        printf("Can't allocate memory\n");
        return;
    }
    MAT_SetElementAll(h_D_Mat, 0.0);
    MAT_PrintVec(h_Z_Vec);
    MAT_PrintVec(h_U_Vec);
    MAT_PrintMat(h_D_Mat);
    ALG_compDistFromEx_Launch(h_Z_Vec, h_U_Vec, h_D_Mat);
    MAT_PrintMat(h_D_Mat);
    free(h_D_Mat.Elements);
}