#include "hip/hip_runtime.h"
//
// Created by Alex on 3/7/2017.
//

#include <stdlib.h>
#include <stdio.h>

#include "alg_initDnun.h"
#include "max_min.h"
#include "mat.h"
#include "dist.h"

/* Private define ------------------------------------------------------------*/
/* Private typedef -----------------------------------------------------------*/
/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
/* Private variables ---------------------------------------------------------*/
/* ---------------------------------------------------------------------------*/

/**
 *@brief  GPU - kernel,
 *@param
 *@retval None
 */
__global__ void ALG_initDnun_Kernel(Tp_Z_Vec_TypeDef Z_Row, Tp_Z_Vec_TypeDef Z_Col, Tp_fMat_TypeDef S_Mat)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Z_Row.Size && col < Z_Col.Size)
    {
        MAT_SetElement(S_Mat, row, col, MAX_MIN_INF);
        if(Z_Row.pElements[row].Label != Z_Col.pElements[col].Label)
        {
            MAT_SetElement(S_Mat, row, col, DIST_Calc_Feat(Z_Row.pElements[row], Z_Col.pElements[col]));
        }
    }
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_initDnun_Launch(const Tp_Z_Vec_TypeDef Z_Row, const Tp_Z_Vec_TypeDef Z_Col, Tp_fVec_TypeDef *pdNUN_Vec)
{
    StopWatchInterface *timer = NULL;

    Tp_Z_Vec_TypeDef d_Z_Row;
    Tp_Z_Vec_TypeDef d_Z_Col;
    Tp_fMat_TypeDef d_S_Mat;


    size_t Size;
    MISC_Bl_Size_TypeDef DimBlck = MISC_Get_Block_Size();

    printf("\nGPU kernel compDistFromEx - Start\n");
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    d_Z_Vec = Z_Vec;
    Size = d_Z_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_Z_Vec.pElements, Size));
    checkCudaErrors(hipMemcpy(d_Z_Vec.pElements, Z_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_U_Vec = U_Vec;
    Size = d_U_Vec.Size * sizeof(int);
    checkCudaErrors(hipMalloc(&d_U_Vec.pElements, Size));
    checkCudaErrors(hipMemcpy(d_U_Vec.pElements, U_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_D_Mat = D_Mat;
    Size = d_D_Mat.Width * d_D_Mat.Height * sizeof(float);
    checkCudaErrors(hipMalloc(&d_D_Mat.Elements, Size));
    checkCudaErrors(hipMemcpy(d_D_Mat.Elements, D_Mat.Elements, Size, hipMemcpyHostToDevice));

    // Invoke kernel
    dim3 dimBlock(DimBlck.Bl_2d, DimBlck.Bl_2d);
    dim3 dimGrid((d_D_Mat.Width + dimBlock.x - 1) / dimBlock.x, (d_D_Mat.Height + dimBlock.y - 1) / dimBlock.y);
    ALG_compDistFromEx_Kernel << < dimGrid, dimBlock >> > (d_Z_Vec, d_U_Vec, d_D_Mat);
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(D_Mat.Elements, d_D_Mat.Elements, Size, hipMemcpyDeviceToHost));

//    Free device memory
    checkCudaErrors(hipFree(d_Z_Vec.pElements));
    checkCudaErrors(hipFree(d_U_Vec.pElements));
    checkCudaErrors(hipFree(d_D_Mat.Elements));

    sdkStopTimer(&timer);
    printf("GPU kernel - Complete, time:%fms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_initDnun_Test(void)
{
    int z_arr[] = {1, 3, 4, 5, 7};
    int u_arr[] = {8, 9, 10};
    Tp_intVec_TypeDef h_Z_Vec;
    Tp_intVec_TypeDef h_U_Vec;
    Tp_fMat_TypeDef h_D_Mat;
    size_t Size;

    h_Z_Vec.pElements = z_arr;
    h_Z_Vec.Size = sizeof(z_arr) / sizeof(z_arr[0]);
    h_U_Vec.pElements = u_arr;
    h_U_Vec.Size = sizeof(u_arr) / sizeof(u_arr[0]);

    h_D_Mat.Height = h_Z_Vec.Size;
    h_D_Mat.Width = h_U_Vec.Size;
    Size = h_D_Mat.Height * h_D_Mat.Width * sizeof(float);
    h_D_Mat.Elements = (float *) malloc(Size);
    if (h_D_Mat.Elements == NULL)
    {
        printf("Can't allocate memory\n");
        return;
    }
    MAT_SetElementAll(h_D_Mat, 0.0);
    MAT_PrintVec(h_Z_Vec);
    MAT_PrintVec(h_U_Vec);
    MAT_PrintMat(h_D_Mat);
    ALG_compDistFromEx_Launch(h_Z_Vec, h_U_Vec, h_D_Mat);
    MAT_PrintMat(h_D_Mat);
    free(h_D_Mat.Elements);
}