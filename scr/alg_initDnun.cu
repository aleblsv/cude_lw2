#include "hip/hip_runtime.h"
//
// Created by Alex on 3/7/2017.
//

#include <stdlib.h>
#include <stdio.h>

#include "alg_initDnun.h"
#include "max_min.h"
#include "mat.h"
#include "dist.h"
#include "misc.h"

/* Private define ------------------------------------------------------------*/
/* Private typedef -----------------------------------------------------------*/
/* Private macro -------------------------------------------------------------*/
/* Private function prototypes -----------------------------------------------*/
/* Private variables ---------------------------------------------------------*/
/* ---------------------------------------------------------------------------*/

/**
 *@brief  GPU - kernel,
 *@param
 *@retval None
 */
__global__ void ALG_initDnun_Kernel(Tp_Z_Vec_TypeDef Z_Row, Tp_Z_Vec_TypeDef Z_Col, Tp_fMat_TypeDef S_Mat)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Z_Row.Size && col < Z_Col.Size)
    {
        MAT_SetElement(S_Mat, row, col, MAX_MIN_INF);
        if(Z_Row.pElements[row].Label != Z_Col.pElements[col].Label)
        {
            MAT_SetElement(S_Mat, row, col, DIST_Calc_Feat(Z_Row.pElements[row], Z_Col.pElements[col]));
        }
    }
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_initDnun_Launch(const Tp_Z_Vec_TypeDef Z_Vec, Tp_fVec_TypeDef *pdNUN_Vec)
{
    StopWatchInterface *timer = NULL;

    Tp_Z_Vec_TypeDef d_Z_Row;
    Tp_Z_Vec_TypeDef d_Z_Col;
    Tp_fMat_TypeDef d_S_Mat;
    Tp_fVec_TypeDef d_dNUN_Vec;
    size_t Size;
    MISC_Bl_Size_TypeDef DimBlck = MISC_Get_Block_Size();

    printf("\nGPU kernel compDistFromEx - Start\n");
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    d_Z_Row = Z_Vec;
    Size = d_Z_Row.Size * sizeof(Tp_Z_TypeDef);
    checkCudaErrors(hipMalloc(&d_Z_Row.pElements, Size));
    checkCudaErrors(hipMemcpy(d_Z_Row.pElements, Z_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_Z_Col = Z_Vec;
    Size = d_Z_Row.Size * sizeof(Tp_Z_TypeDef);
    checkCudaErrors(hipMalloc(&d_Z_Row.pElements, Size));
    checkCudaErrors(hipMemcpy(d_Z_Row.pElements, Z_Vec.pElements, Size, hipMemcpyHostToDevice));

    d_S_Mat.Width = d_Z_Col.Size;
    d_S_Mat.Height = d_Z_Row.Size;
    Size = d_S_Mat.Width * d_S_Mat.Height * sizeof(float);
    checkCudaErrors(hipMalloc(&d_S_Mat.pElements, Size));

    // Invoke kernel
    dim3 dimBlock(DimBlck.Bl_2d, DimBlck.Bl_2d);
    dim3 dimGrid((d_S_Mat.Width + dimBlock.x - 1) / dimBlock.x, (d_S_Mat.Height + dimBlock.y - 1) / dimBlock.y);
    ALG_initDnun_Kernel << < dimGrid, dimBlock >> > (d_Z_Row, d_Z_Col, d_S_Mat);
    hipDeviceSynchronize();

    MAT_PrintMat(d_S_Mat);
    //ToDo:

    checkCudaErrors(hipMemcpy(pdNUN_Vec, d_dNUN_Vec, Size, hipMemcpyDeviceToHost));

//    Free device memory
    checkCudaErrors(hipFree(d_Z_Row.pElements));
    checkCudaErrors(hipFree(d_Z_Col.pElements));
    checkCudaErrors(hipFree(d_S_Mat.pElements));
    checkCudaErrors(hipFree(d_dNUN_Vec.pElements));

    sdkStopTimer(&timer);
    printf("GPU kernel - Complete, time:%fms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

/**
 *@brief
 *@param
 *@retval None
 */
void ALG_initDnun_Test(void)
{
    float feat1_arr[] = {2.0, 3.0};
    float feat2_arr[] = {1.0, 5.0};
    Tp_Z_TypeDef z_arr[] = {
            {(sizeof(feat1_arr), feat1_arr, 1, 0)},
            {(sizeof(feat2_arr), feat2_arr, 2, 0)}
    };
    Tp_Z_Vec_TypeDef Z_Vec;

    Z_Vec.Size = sizeof(z_arr) / sizeof(z_arr[0]);
    Z_Vec.pElements = z_arr;

    if (h_D_Mat.pElements == NULL)
    {
        printf("Can't allocate memory\n");
        return;
    }
    MAT_SetElementAll(h_D_Mat, 0.0);
    MAT_PrintVec(h_Z_Vec);
    MAT_PrintVec(h_U_Vec);
    MAT_PrintMat(h_D_Mat);
    ALG_compDistFromEx_Launch(h_Z_Vec, h_U_Vec, h_D_Mat);
    MAT_PrintMat(h_D_Mat);
    free(h_D_Mat.pElements);
}